#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <malloc.h>
#include <time.h>

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#define max(x, y) ((x) > (y) ? (x) : (y))


__global__ void restore(double* mas, int N){
    size_t i = threadIdx.x;
    double shag = 10.0 / (N-1);
    if (i < N - 1 && i > 0){
        mas[i] = 10.0 + i * shag;
        mas[i * N] = 10.0 + i * shag;
        mas[N - 1 + i * N] = 20.0 + i * shag;
        mas[N * (N - 1) + i] = 20.0 + i * shag;
    }
}


__global__ void updateTemperature(const double *arr_pred, double *arr_new, size_t N){
    int i = blockIdx.x;
    int j = threadIdx.x; 
    if (i < N - 1 && j < N - 1 && i > 0 && j < 0) {
        arr_new[i * N + j] = 0.25 * (arr_pred[i*N+j-1] + arr_pred[(i - 1) * N + j] +
                                 arr_pred[(i+1)*N+j] + arr_pred[i * N + j + 1]);
    }
}


__global__ void update_matrix(const double* arr_pred, double* arr_new){
    int i = blockIdx.x * blockDim.x + threadIdx.x; //вычисления линейного индекса элемента внутри сетки CUDA
    if (i < N - 1 && i > 0){
        arr_new[i] = arr_pred[i] - arr_new[i];
    }
}


int main(int argc, char* argv[]) {
    clock_t a = clock();
    int size;
    double tol;
    int iter_max;
    if (argc < 4){
        printf("Неправильное количество аргументов");
        exit(1);
    }
    tol = strtod(argv[1], NULL);
    if (tol <= 0){
        printf("Ограничение точности должно превышать 0");
        exit(1);
    }
    size = atoi(argv[2]);
    if (size <= 0){
        printf("Размер матриццы должен быть больше 0");
        exit(1);
    }
    iter_max = atoi(argv[3]);
    if (iter_max <= 0){
        printf("Максимальное количество итераци должно быть больше 0");
        exit(1);
    }

    hipSetDevice(0);

    int num_iter = 0;
    double error = 1.0;

    hipStream_t stream; // указатель на объект потока CUDA
    hipStreamCreate(&stream); // создание потока CUDA

    hipGraph_t graph; //указатель на объект графа CUDA
    hipGraphExec_t graph_exec; // указатель на объект выполнения графа CUDA

    double *arr_pred, *arr_new;
    hipMalloc((void **)&arr_pred, sizeof(double) * size * size);
    hipMalloc((void **)&arr_new, sizeof(double) * size * size);
    
    // Выделение памяти на хосте
    double* host_arr_pred = (double*)malloc(sizeof(double) * size * size);

    // Заполнение границ массива
    double shag = 10.0 / (size - 1);
    for (size_t i = 0; i < size; i++) {
        host_arr_pred[i] = 10.0 + i * shag;
        host_arr_pred[i * size] = 10.0 + i * shag;
        host_arr_pred[size - 1 + i * size] = 20.0 + i * shag;
        host_arr_pred[size * (size - 1) + i] = 20.0 + i * shag;
    }

    // Копирование данных из хоста в устройство
    hipMemcpy(arr_pred, host_arr_pred, sizeof(double) * size * size, hipMemcpyHostToDevice);
    free(host_arr_pred); // Освобождение памяти на хосте
    
    // копирование данных из хоста на устройство
    hipMemcpy(arr_new, arr_pred, sizeof(double) * size * size, hipMemcpyHostToDevice);

    // выделяем память на gpu. Хранение ошибки на device
    double *mas_error = 0;
    hipMalloc((void **)&mas_error, sizeof(double)); //выделение памяти для GPU

    size_t tempStorageBytes = 0;
    double *tempStorage = NULL; // временного хранения буфера для операции редукции на GPU
    
   dim3 thread = size < 1024 ? size : 1024;
    dim3 block = size / (size < 1024 ? size : 1024);

    // получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, arr_new, mas_error, size * size, stream);

    hipMalloc(&tempStorage, tempStorageBytes); //выделение памяти для буфера
    //
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal); //записывает операции, выполняемые в потоке

    for (size_t i = 0; i < 100; i += 2) {
        updateTemperature<<<block, thread, 0, stream>>>(arr_pred, arr_new, size); // количество потоков в блоке, количество блоков, разделяемая память
        updateTemperature<<<block, thread, 0, stream>>>(arr_new, arr_pred, size);
    }
            
    update_matrix<<<block, thread, 0, stream>>>(arr_pred, arr_new);

    hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, arr_new, mas_error, size * size, stream);
    restore<<<1, size, 0, stream>>>(arr_new, size);
        

    hipStreamEndCapture(stream, &graph); //завершение захвата операций    
    hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0); // создание граф выполнения
       

    while ((iter_max > num_iter) && (error > tol)) {
        hipGraphLaunch(graph_exec, stream); // его запуск
        hipStreamSynchronize(stream);
        num_iter+=100;
        
        printf("%d : %lf\n", num_iter, error);
        fflush(stdout); //  проверить, что все данные, которые были записаны в буфер вывода с помощью функции printf(), записались
    }


    printf("Финальные результаты: %d, %0.6lf\n", num_iter, error);
   
   // Копирование данных из устройства на хост
   double* host_arr_pred_1 = (double*)malloc(sizeof(double) * size * size);
   hipMemcpy(host_arr_pred_1, arr_pred, sizeof(double) * size * size, hipMemcpyDeviceToHost);

   // Вывод матрицы на экран
   printf("Матрица arr_pred после выполнения операций:\n");
   for (int i = 0; i < size; i++) {
      for (int j = 0; j < size; j++) {
         printf("%0.2lf ", host_arr_pred_1[i * size + j]);
      }
      printf("\n");
   }

   free(host_arr_pred_1); // Освобождение памяти на хосте

    hipStreamDestroy(stream);
    hipGraphDestroy(graph);

    hipFree(arr_pred);
    hipFree(arr_new);

    clock_t b=clock();
    double d=(double)(b-a)/CLOCKS_PER_SEC; // переводит в секунды
    printf("%.25f время в секундах", d);

    return 0;
}
