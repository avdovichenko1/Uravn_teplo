#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "mpi.h"

// функция изменения матрицы уравнения теплопроводности
__global__ void calculate(double *CudaArr, double *CudaNewArr, size_t MatrixX, size_t MatrixY)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x; //вычисления линейного индекса элемента внутри сетки 
    size_t j =  blockDim.y * blockIdx.y + threadIdx.y; 
    int index = i * MatrixX + j;
    if ((i < MatrixX - 1 && j < MatrixY - 1 && i > 0 && j > 0)) 
        CudaNewArr[index] = 0.25 * (CudaArr[(i - 1) * MatrixX + j] + CudaArr[(i + 1) * MatrixX + j] + CudaArr[index - 1] + CudaArr[index + 1]);
}


// функция разницы матриц
__global__ void subtraction(double* CudaArr, double* CudaNewArr, double* CudaArrErr, size_t Matrix)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;  
    size_t j =  blockDim.y * blockIdx.y + threadIdx.y;
    int idx = i * Matrix + j; 
    if ((i < Matrix && j < Matrix && i > 0 && j > 0))
	    CudaArrErr[idx] = CudaArr[idx] - CudaNewArr[idx];
}

// функция востановления границ матрицы
__global__ void restore(double* arr, int size){
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i >= size-1)
        	return;
	
	arr[i] = 10.0 + i * 10.0 / (size - 1);
	arr[i * size] = 10.0 + i * 10.0 / (size - 1);
	arr[size - 1 + i * size] = 20.0 + i * 10.0 / (size - 1);
	arr[size * (size - 1) + i] = 20.0 + i * 10.0 / (size - 1);
}

int find_threads(int size){
	if (size%32==0)
		return size/1024;

	return int(size/1024)+1;

}

int main(int argc, char* argv[]) {
    
    double time_spent = 0.0;
    clock_t begin = clock(); 

    // Convert command line arguments to integers
    int Matrix = atoi(argv[1]);
    double accuracy = atof(argv[2]);
    int iterations = atoi(argv[3]);

    int rank, size;
    /* Initialize the MPI library */
    MPI_Init(&argc,&argv);
    /* Determine the calling process rank and total number of ranks */
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    MPI_Comm_size(MPI_COMM_WORLD,&size);
    /* Call MPI routines like MPI_Send, MPI_Recv, ... */
    hipSetDevice(rank);       

    if (rank != 0)
        hipDeviceEnablePeerAccess(rank - 1, 0);
    if (rank != (size-1))
        hipDeviceEnablePeerAccess(rank + 1, 0);
	
    size_t size_y = Matrix / size + 1;
    if (rank != size - 1 && rank != 0) 
	    size_y += 1;
	
    dim3 t(32,32); //определяю количество нитей в каждом блоке
    dim3 b(find_threads(Matrix), find_threads(Matrix)); // количество блоков
	
    // выделяем память на gpu через cuda для 3 сеток
    double *A, *CudaArr, *CudaNewArr, *CudaArrErr;
    hipMalloc((void **)&CudaArr, sizeof(double) * Matrix * size_y);
    hipMalloc((void **)&CudaNewArr, sizeof(double) * Matrix * size_y);
    hipMalloc((void **)&CudaArrErr, sizeof(double) * Matrix * size_y);

    hipHostMalloc(&A, sizeof(double) * Matrix * Matrix);
    restore<<<b, t>>>(A, Matrix);
	
    size_t offset = (rank != 0) ? Matrix : 0;
    hipMemcpy(CudaArr, A + (Matrix * Matrix * rank / size) - offset, sizeof(double) * Matrix * size_y, hipMemcpyHostToDevice);
    hipMemcpy(CudaNewArr, A + (Matrix * Matrix * rank / size) - offset, sizeof(double) * Matrix * size_y, hipMemcpyHostToDevice);

    // выделяем память на gpu. Хранение ошибки на device
    double *max_err = 0;
    hipMalloc((void **)&max_err, sizeof(double));

    size_t tempStorageBytes = 0;
    double *tempStorage = NULL;

    // получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, CudaNewArr, max_err, Matrix * size_y);

    // выделяем память для буфера
    hipMalloc(&tempStorage, tempStorageBytes);

    hipStream_t stream;
    hipStreamCreate(&stream);	

    // Main loop
    double err = 1;
    int iter = 0;

    while (err > accuracy && iter < iterations) 
    {
	iter++;
	calculate <<<b, t, 0, stream>>> (CudaArr, CudaNewArr, Matrix, size_y);
	// Расчитываем ошибку каждую сотую итерацию
	if (iter % 100 == 0) {
		subtraction<<<b, t, 0, stream>>>(CudaArr, CudaNewArr, CudaArrErr, Matrix);
		hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, CudaArrErr, max_err, Matrix * size_y);
		hipMemcpy(&err, max_err, sizeof(double), hipMemcpyDeviceToHost);

		MPI_Allreduce((void*)&max_err,(void*)&max_err, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
		
		hipMemcpyAsync(&err, max_err, sizeof(double), hipMemcpyDeviceToHost, stream); // запись ошибки в переменную на host
            	// Находим максимальную ошибку среди всех и передаём её всем процессам
	}

	if (rank != 0){ // Обмен верхней границей
            	MPI_Sendrecv(CudaNewArr + Matrix + 1, Matrix - 2, MPI_DOUBLE, rank - 1, 0, CudaNewArr + 1, 
			     Matrix - 2, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	}	
        if (rank != size - 1) { // Обмен нижней границей
            	MPI_Sendrecv(CudaNewArr + (size_y - 2) * Matrix + 1, Matrix - 2, MPI_DOUBLE, rank + 1, 0, 
			     CudaNewArr + (size_y - 1) * Matrix + 1, Matrix - 2, MPI_DOUBLE, rank + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	}    
	double* c = CudaNewArr; 
        CudaNewArr = CudaArr;
        CudaArr = c;
    }

    printf("Final result: %d, %0.6lf\n", iter, err);


    hipFree(CudaArr);
    hipFree(CudaNewArr);
    hipFree(CudaArrErr);
    hipFree(A);

    MPI_Finalize();
	
    clock_t end = clock();
    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time elapsed: %f\n", time_spent);

    return 0;
}
