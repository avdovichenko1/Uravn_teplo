#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <malloc.h>
#include <time.h>

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#define max(x, y) ((x) > (y) ? (x) : (y))


__global__ void updateTemperature(const double *arr_pred, double *arr_new, size_t N)
{
    int i = blockIdx.x + 1; // размер строки
    int j = threadIdx.x + 1; // столбца
    arr_new[i * N + j] = 0.25 * (arr_pred[i*N+j-1] + arr_pred[(i - 1) * N + j] +
                                 arr_pred[(i+1)*N+j] + arr_pred[i * N + j + 1]);
}


__global__ void update_matrix(const double* arr_pred, double* arr_new)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    arr_new[i] = arr_pred[i] - arr_new[i];
}

// Функция востановления границ матрицы
__global__ void restore(double* mas, int N){
    size_t i = threadIdx.x;
    double shag = 10.0 / (N-1);
    mas[i] = 10.0 + i * shag;
    mas[i * N] = 10.0 + i * shag;
    mas[N - 1 + i * N] = 20.0 + i * shag;
    mas[N * (N - 1) + i] = 20.0 + i * shag;
}

int main(int argc, char* argv[]) {
    clock_t a = clock();
    int size;
    double tol;
    int iter_max;
    if (argc < 4){
        printf("Неправильное количество аргументов");
        exit(1);
    }
    tol = strtod(argv[1], NULL);
    if (tol <= 0){
        printf("Ограничение точности должно превышать 0");
        exit(1);
    }
    size = atoi(argv[2]);
    if (size <= 0){
        printf("Размер матриццы должен быть больше 0");
        exit(1);
    }
    iter_max = atoi(argv[3]);
    if (iter_max <= 0){
        printf("Максимальное количество итераци должно быть больше 0");
        exit(1);
    }

    hipSetDevice(1);

    int num_iter = 0;
    double error = 1.0;

    hipStream_t stream; // указатель на объект потока CUDA
    hipStreamCreate(&stream); // создание потока CUDA

    hipGraph_t graph; //указатель на объект графа CUDA
    hipGraphExec_t graph_exec; // указатель на объект выполнения графа CUDA

    double *arr_pred, *arr_new;
    hipMalloc((void **)&arr_pred, sizeof(double) * size * size);
    hipMalloc((void **)&arr_new, sizeof(double) * size * size);
    
    // Функция востановления границ матрицы
    {
        int i = threadIdx.x;
        double shag = 10.0 / (size-1);
        arr_pred[i] = 10.0 + i * shag;
        arr_pred[i * size] = 10.0 + i * shag;
        arr_pred[size - 1 + i * size] = 20.0 + i * shag;
        arr_pred[size * (size - 1) + i] = 20.0 + i * shag;
    }

    //restore<<<1, size>>>(arr_pred, size); //заполнение массива
    // копирование данных из хоста на устройство
    hipMemcpy(arr_new, arr_pred, sizeof(double) * size * size, hipMemcpyHostToDevice);

    // выделяем память на gpu. Хранение ошибки на device
    double *mas_error = 0;
    hipMalloc((void **)&mas_error, sizeof(double)); //выделение памяти для GPU

    size_t tempStorageBytes = 0;
    double *tempStorage = NULL; // временного хранения буфера для операции редукции на GPU

    // получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, arr_new, mas_error, size * size, stream);

    hipMalloc(&tempStorage, tempStorageBytes); //выделение памяти для буфера

    bool graphCreated = false;

    while ((iter_max > num_iter) && (error > tol)) {
        if(!graphCreated){
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

            for (size_t i = 0; i < 100; i += 2) {
                updateTemperature<<<size - 2, size - 2, 0, stream>>>(arr_pred, arr_new, size);
                updateTemperature<<<size - 2, size - 2, 0, stream>>>(arr_new, arr_pred, size);
            }
            update_matrix<<<size, size, 0, stream>>>(arr_pred, arr_new);

            hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, arr_new, mas_error, size * size, stream);
            restore<<<1, size, 0, stream>>>(arr_new, size);

            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);
            graphCreated=true;

        }
        else{
            hipGraphLaunch(graph_exec, stream);
            hipMemcpyAsync(&error, mas_error, sizeof(double), hipMemcpyDeviceToHost, stream);
            hipStreamSynchronize(stream);
            num_iter+=100;
            graphCreated=false;
        }

    }

    printf("Финальные результаты: %d, %0.6lf\n", num_iter, error);

    // удаление потока и графа
    hipStreamDestroy(stream);
    hipGraphDestroy(graph);

    hipFree(arr_pred);
    hipFree(arr_new);

    clock_t b=clock();
    double d=(double)(b-a)/CLOCKS_PER_SEC; // переводит в секунды
    printf("%.25f время в секундах", d);

    return 0;
}
